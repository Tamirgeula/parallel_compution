#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>

#include<stdio.h>
#include"K-means_structs.h"
#include "constants.h"
#ifndef __cudaSection
#define __cudaSection
extern "C" unsigned int cudaMaxThredsPerBlock() {
	int nDevices;
	hipGetDeviceCount(&nDevices);

	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	return prop.maxThreadsDim[0];

}





//__global__ void cudaFindMaxArr(double * pointsDis, unsigned int*  numOfPoints) {// logN finding max insted of N
//	__shared__ double tempNumOfPoints;
//	tempNumOfPoints = *numOfPoints;
//	double maxDis = 0;
//	unsigned int threadID = blockIdx.x*blockDim.x + threadIdx.x;
//	while (threadID<tempNumOfPoints) {
//		if (threadID % 2 == 0 && threadID != tempNumOfPoints - 1) {
//			maxDis = (pointsDis[threadID] > pointsDis[threadID + 1]) ? pointsDis[threadID] : pointsDis[threadID + 1];
//			pointsDis[threadID / 2] = maxDis;
//		}
//		__syncthreads();
//
//		if (threadID == tempNumOfPoints - 1) {
//			maxDis = (pointsDis[threadID] > pointsDis[threadID - 1]) ? pointsDis[threadID] : pointsDis[threadID - 1];
//			unsigned int index = ((int)tempNumOfPoints % 2 == 1) ? (threadID / 2) : ((threadID - 1) / 2);
//			pointsDis[index] = maxDis;
//		}
//
//
//		tempNumOfPoints = ceil(tempNumOfPoints / 2);
//
//		__syncthreads();
//		if (tempNumOfPoints == 1)
//			break;
//	}
//}
//extern "C" void cudaFindMaxArrMediator(double * arr, unsigned int  arrSize, double* max) {
//	double *d_arr;
//	unsigned  int* d_arrSize;
//
//	hipMalloc(&d_arr, arrSize * sizeof(double));
//	hipMalloc(&d_arrSize, sizeof(unsigned int));
//
//	hipMemcpy(d_arr, arr, arrSize * sizeof(double), hipMemcpyHostToDevice);
//	hipMemcpy(d_arrSize, &arrSize, sizeof(unsigned int), hipMemcpyHostToDevice);
//
//	unsigned int numOfBlocks = (unsigned int)ceil(arrSize*1.0 / cudaMaxThredsPerBlock());
//	unsigned int blocksPerThreds = (numOfBlocks>1) ? cudaMaxThredsPerBlock() : arrSize;
//	cudaFindMaxArr << <numOfBlocks, blocksPerThreds >> > (d_arr, d_arrSize);
//
//	hipMemcpy(max, &d_arr[0], sizeof(double), hipMemcpyDeviceToHost);//update point
//
//	hipFree(d_arr);
//	hipFree(d_arrSize);
//
//}


__global__ void cudaCalcPoints(point* points, unsigned int*  numOfPoints, double* t) {

	unsigned int threadID = blockIdx.x*blockDim.x + threadIdx.x;

	if (threadID <*numOfPoints) {
		points[threadID].x = points[threadID].xi + points[threadID].vxi*(*t);
		points[threadID].y = points[threadID].yi + points[threadID].vyi*(*t);
		points[threadID].z = points[threadID].zi + points[threadID].vzi*(*t);

	}



}
extern "C" void cudaCalcPointsMediator(point *points, unsigned  int numOfPoints, double t) {
	point *d_points;
	unsigned  int* d_numOfPoints;
	double* d_t;

	hipMalloc(&d_points, numOfPoints * sizeof(point));
	hipMalloc(&d_numOfPoints, sizeof(unsigned int));
	hipMalloc(&d_t, sizeof(double));

	hipMemcpy(d_points, points, numOfPoints * sizeof(point), hipMemcpyHostToDevice);
	hipMemcpy(d_numOfPoints, &numOfPoints, sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(d_t, &t, sizeof(double), hipMemcpyHostToDevice);


	unsigned int numOfBlocks = (unsigned int)ceil(numOfPoints*1.0 / cudaMaxThredsPerBlock());
	unsigned int blocksPerThreds = (numOfBlocks>1) ? cudaMaxThredsPerBlock() : numOfPoints;
	cudaCalcPoints << <numOfBlocks, blocksPerThreds >> > (d_points, d_numOfPoints, d_t);//calc the GPU 2 qurter

	hipMemcpy(points, d_points, numOfPoints * sizeof(point), hipMemcpyDeviceToHost);//update point

	hipFree(d_points);
	hipFree(d_numOfPoints);
	hipFree(d_t);
}

//__global__ void cudaCalcPointsDis(point *points, unsigned  int* numOfPoints, double * distances) {
//	unsigned int threadID = blockIdx.x*blockDim.x + threadIdx.x;
//	if (threadID != 0 && threadID < *numOfPoints) {//threadID != 0 because i don't want to calc dis of point from itself
//		double dis=cudaCalcEuclideanDistancePP(&points[0],&points[threadID]);
//		distances[threadID - 1] = dis;
//	}
//
//}

//extern "C" void cudaCalcPointsMaxDisMediator(point *points, unsigned  int numOfPoints,double* maxDis) {
//
//	if (numOfPoints == 2) {
//		*maxDis = euclideanDistancePP(&points[0], &points[1]);
//		return;
//	}
//	
//
//	point *d_points;
//	unsigned  int* d_numOfPoints;
//	double *d_distances;
//
//	hipMalloc(&d_points, numOfPoints * sizeof(point));
//	hipMalloc(&d_numOfPoints, sizeof(unsigned int));
//	hipMalloc(&d_distances, sizeof(double)*(numOfPoints - 1));
//
//
//	hipMemcpy(d_points, points, numOfPoints * sizeof(point), hipMemcpyHostToDevice);
//	hipMemcpy(d_numOfPoints, &numOfPoints, sizeof(unsigned int), hipMemcpyHostToDevice);
//
//
//	unsigned int numOfBlocks = (unsigned int)ceil(numOfPoints*1.0 / cudaMaxThredsPerBlock());
//	unsigned int blocksPerThreds = (numOfBlocks>1) ? cudaMaxThredsPerBlock() : numOfPoints;
//
//
//	cudaCalcPointsDis << <numOfBlocks, blocksPerThreds >> > (d_points, d_numOfPoints, d_distances);////calc in O(1) dis of point[i] with all the points ahead
//
//
//	
//
//	numOfPoints--;
//	hipMemcpy(d_numOfPoints, &numOfPoints, sizeof(unsigned int), hipMemcpyHostToDevice);
//	cudaFindMaxArr << <numOfBlocks, blocksPerThreds >> > (d_distances, d_numOfPoints);//log(n)
//
//	hipMemcpy(maxDis, &d_distances[0], sizeof(double), hipMemcpyDeviceToHost);
//
//
//	hipFree(d_points);
//	hipFree(d_numOfPoints);
//	hipFree(d_distances);
//
//}



//__global__ void  cudaSortPointsToGroups(point * points, unsigned int* numOfPoints, centroid* centroids, point ** pointsGropByCentoids) {
//	unsigned int threadID = blockIdx.x*blockDim.x + threadIdx.x;
//
//	if (threadID < *numOfPoints) {
//
//		pointsGropByCentoids[points[threadID].closestCentroidIndex][points[threadID].index%]
//	
//	}
//}
//extern "C" point ** cudaSortPointsToGroupsMediator(point * points, unsigned int numOfPoints, centroid* centroids, unsigned int numOfcentroids) {
//
//
//	point *d_points,**d_pointsGropByCentoids;
//	centroid * d_centroids;
//	unsigned  int* d_numOfPoints;
//
//	hipMalloc(&d_points, numOfPoints * sizeof(point));
//	hipMalloc(&d_numOfPoints, sizeof(unsigned int));
//	hipMalloc(&d_pointsGropByCentoids, sizeof(point*)*numOfcentroids);
//	hipMalloc(&d_centroids, sizeof(centroid)*numOfcentroids);
//
//	for (unsigned int i = 0; i < numOfcentroids; i++){
//		hipMalloc(&d_pointsGropByCentoids[i], sizeof(point)*centroids[i].numOfPoints);
//	}
//
//
//	hipMemcpy(d_points, points, numOfPoints * sizeof(point), hipMemcpyHostToDevice);
//	hipMemcpy(d_numOfPoints, &numOfPoints, sizeof(unsigned int), hipMemcpyHostToDevice);
//	hipMemcpy(d_centroids, centroids, sizeof(centroid)*numOfcentroids, hipMemcpyHostToDevice);
//
//
//	unsigned int numOfBlocks = (unsigned int)ceil(numOfPoints*1.0 / cudaMaxThredsPerBlock());
//	unsigned int blocksPerThreds = (numOfBlocks>1) ? cudaMaxThredsPerBlock() : numOfPoints;
//	cudaSortPointsToGroups << <numOfBlocks, blocksPerThreds >> > (d_points, d_numOfPoints, d_centroids, d_pointsGropByCentoids);//calc the GPU 2 qurter
//
//}
//__global__ void cudaCalcMaxPointDis(point* points, unsigned int*  numOfPoints,double * pointsDis) {
//
//	unsigned int threadID = blockIdx.x*blockDim.x + threadIdx.x;
//	double dis = 0;
//	if(threadID<*numOfPoints)
//	for (unsigned int i = 0; i < *numOfPoints; i++){
//		if (threadID > i) {
//			dis = cudaCalcEuclideanDistancePP(&points[i], &points[threadID]);
//			pointsDis[threadID] = dis;
//		}
//		
//		__syncthreads();
//		if (threadID == 0) {
//			double localMax;
//			//cudaFindMaxArr<<<>>>(pointsDis, numOfPoints);
//			if (localMax > globalMaxDis)
//				globalMaxDis = localMax;
//		}
//		__syncthreads();
//
//	}
//
//		
//}


//__device__ double cudaCalcEuclideanDistancePP(point * p1, point* p2) {
//	double distance = sqrt(pow(p1->x - p2->x, 2) + pow(p1->y - p2->y, 2) + pow(p1->z - p2->z, 2));
//	return distance;
//}











extern "C" void printCudaDitels() {
	int nDevices;
	hipGetDeviceCount(&nDevices);
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		//printf("Device Number: %d\n", i);
		//printf("Device name: %s\n", prop.name);
		//printf("Memory Clock Rate (KHz): %d\n",
		//	prop.memoryClockRate);
		//printf("Memory Bus Width (bits): %d\n",
		//	prop.memoryBusWidth);
		//printf("Peak Memory Bandwidth (GB/s): %f\n",
		//	2.0*prop.memoryClockRate*(prop.memoryBusWidth / 8) / 1.0e6);
		printf("Num of multi processor:%d\n", prop.multiProcessorCount);
		printf("Max Threads Per Multi Processor:%d\n", prop.maxThreadsPerMultiProcessor);
		printf("\n");

		printf("Max size of each dim of grid(%d,%d,%d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("Max thread Dim (%d,%d,%d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("Max threads per block:%d\n", prop.maxThreadsPerBlock);

		printf("Max texture 3D:(%d,%d,%d)\n", prop.maxTexture3D[0], prop.maxTexture3D[1], prop.maxTexture3D[2]);
		printf("Max texture 2D:(%d,%d)\n", prop.maxTexture2D[0], prop.maxTexture2D[1]);

	}

}





#endif // !__cudaSection



